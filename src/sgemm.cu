#include "hip/hip_runtime.h"
#include "runner.cuh"
#include "matrix_utils.cuh"

#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>

#define KernelErrChk()                                                        \
    {                                                                         \
        hipError_t errSync = hipGetLastError();                             \
        hipError_t errAsync = hipDeviceSynchronize();                       \
        if (errSync != hipSuccess)                                           \
        {                                                                     \
            printf("Sync kernel error: %s\n", hipGetErrorString(errSync));   \
            exit(EXIT_FAILURE);                                               \
        }                                                                     \
        if (errAsync != hipSuccess)                                          \
        {                                                                     \
            printf("Async kernel error: %s\n", hipGetErrorString(errAsync)); \
            exit(EXIT_FAILURE);                                               \
        }                                                                     \
    }

bool run_kernel(float *A,
                float *B,
                float *C,
                int m,
                int n,
                int k,
                size_t pitch_A = 0,
                size_t pitch_B = 0,
                size_t pitch_C = 0,
                int run_type = 0)
{
    switch (run_type)
    {
    case 0:
        run_sgemm_naive(A, B, C, m, n, k);
        return true;
    case 1:
        run_sgemm_global_memory_coalescing(A, B, C, m, n, k);
        return true;
    case 2:
        run_sgemm_shared_memory(A, B, C, m, n, k);
        return true;
    case 3:
        run_sgemm_blocktiling_1d(A, B, C, m, n, k);
        return true;
    case 4:
        run_sgemm_blocktiling_2d(A, B, C, m, n, k);
        return true;
    case 5:
        run_sgemm_vectorize(A, B, C, m, n, k);
        return true;
    case 6:
        run_sgemm_vectorize_v2(A, B, C, m, n, k, pitch_A, pitch_B, pitch_C);
        return true;
    default:
        printf("Invalid run type\n");
        return false;
    }
}

int main(int argc, char **argv)
{
    int m = atoi(argv[1]);
    int n = atoi(argv[2]);
    int k = atoi(argv[3]);
    int run_type = atoi(argv[4]);

    // Allocate memory for matrices
    float *A, *B, *C, *C_ref;
    float *d_A, *d_B, *d_C, *d_C_ref;
    // for pitch version
    float *d_A_p, *d_B_p, *d_C_p;
    size_t pitch_A, pitch_B, pitch_C;

    A = new float[m * k];
    B = new float[k * n];
    C = new float[m * n];
    // save reference result
    C_ref = new float[m * n];

    // Initialize matrices: use range_init_matrix/randomize_matrix/zero_init_matrix
    randomize_matrix(A, m * k);
    randomize_matrix(B, k * n);
    zero_init_matrix(C, m * n);
    zero_init_matrix(C_ref, m * n);

    hipMalloc((void **)&d_A, m * k * sizeof(float));
    hipMalloc((void **)&d_B, k * n * sizeof(float));
    hipMalloc((void **)&d_C, m * n * sizeof(float));
    hipMalloc((void **)&d_C_ref, m * n * sizeof(float));

    // Copy matrices to device
    hipMemcpy(d_A, A, m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, k * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C_ref, C_ref, m * n * sizeof(float), hipMemcpyHostToDevice);

    if (run_type >= 6)
    {
        // use hipMallocPitch to allocate memory for matrices
        hipMallocPitch((void **)&d_A_p, &pitch_A, k * sizeof(float), m);
        hipMallocPitch((void **)&d_B_p, &pitch_B, n * sizeof(float), k);
        hipMallocPitch((void **)&d_C_p, &pitch_C, n * sizeof(float), m);
        // Copy matrices to device
        hipMemcpy2D(d_A_p, pitch_A, A, k * sizeof(float), k * sizeof(float), m, hipMemcpyHostToDevice);
        hipMemcpy2D(d_B_p, pitch_B, B, n * sizeof(float), n * sizeof(float), k, hipMemcpyHostToDevice);
        hipMemcpy2D(d_C_p, pitch_C, C, n * sizeof(float), n * sizeof(float), m, hipMemcpyHostToDevice);
    }

    // Run reference matrix multiplication
    run_cutlass_sgemm(d_A, d_B, d_C_ref, m, n, k);

    hipDeviceSynchronize();
    KernelErrChk();

    // Run matrix multiplication
    bool run_success = false;
    if (run_type >= 6)
    {
        run_success =
            run_kernel(d_A_p, d_B_p, d_C_p, m, n, k, pitch_A, pitch_B, pitch_C, run_type);
    }
    else
    {
        run_success =
            run_kernel(d_A, d_B, d_C, m, n, k, pitch_A, pitch_B, pitch_C, run_type);
    }

    if (!run_success)
    {
        std::cout << "Invalid run type" << std::endl;
        return 0;
    }
    hipDeviceSynchronize();
    KernelErrChk();

    // Copy result back to host
    if (run_type >= 6)
    {
        hipMemcpy2D(
            C,
            n * sizeof(float),
            d_C_p,
            pitch_C,
            n * sizeof(float),
            m,
            hipMemcpyDeviceToHost);
    }
    else
    {
        hipMemcpy(C, d_C, m * n * sizeof(float), hipMemcpyDeviceToHost);
    }
    hipMemcpy(C_ref, d_C_ref, m * n * sizeof(float), hipMemcpyDeviceToHost);

    // Check result
    bool correct = true;
    float eps = 1e-6;
    for (int i = 0; i < m * n; i++)
    {
        if (abs(C[i] - C_ref[i]) > eps)
        {
            printf("Error at position %d, expected %f, get %f\n", i, C_ref[i], C[i]);
            correct = false;
            break;
        }
    }

    if (correct)
    {
        // run speed test
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
        for (int i = 0; i < 100; i++)
        {
            if (run_type >= 6)
            {
                run_kernel(d_A_p, d_B_p, d_C_p, m, n, k, pitch_A, pitch_B, pitch_C, run_type);
            }
            else 
            {
                run_kernel(d_A, d_B, d_C, m, n, k, pitch_A, pitch_B, pitch_C, run_type);
            }
            hipDeviceSynchronize();
        }
        KernelErrChk();
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float elapsed_time = 0.0;
        hipEventElapsedTime(&elapsed_time, start, stop);
        float avg_run_time = elapsed_time * 1000 / 100;
        std::cout << "Average run time: " << avg_run_time << " us" << std::endl;
    }

    // Free memory
    delete[] A;
    delete[] B;
    delete[] C;
    delete[] C_ref;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_C_ref);
}